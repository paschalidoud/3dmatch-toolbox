#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <sstream>
#include <string>
#include "marvin.hpp"

#define CUDA_NUM_THREADS 512
#define CUDA_MAX_NUM_BLOCKS 2880

#define IS_PLY_BINARY false

struct tdf_struct {
    float origin_x;
    float origin_y;
    float origin_z;
    int dim_x;
    int dim_y;
    int dim_z;
    float * tdf_values;
};

int random_number(int max_value, int min_value) {
    return rand() % max_value + min_value;
}

// CUDA kernel function to compute TDF voxel grid values given a point cloud (warning: approximate, but fast)
__global__
void ComputeTDF(int CUDA_LOOP_IDX, float * voxel_grid_occ, float * voxel_grid_TDF,
                int voxel_grid_dim_x, int voxel_grid_dim_y, int voxel_grid_dim_z,
                float voxel_grid_origin_x, float voxel_grid_origin_y, float voxel_grid_origin_z,
                float voxel_size, float trunc_margin) {

  int voxel_idx = CUDA_LOOP_IDX * CUDA_NUM_THREADS * CUDA_MAX_NUM_BLOCKS + blockIdx.x * CUDA_NUM_THREADS + threadIdx.x;
  if (voxel_idx > (voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z))
    return;

  int pt_grid_z = (int)floor((float)voxel_idx / ((float)voxel_grid_dim_x * (float)voxel_grid_dim_y));
  int pt_grid_y = (int)floor(((float)voxel_idx - ((float)pt_grid_z * (float)voxel_grid_dim_x * (float)voxel_grid_dim_y)) / (float)voxel_grid_dim_x);
  int pt_grid_x = (int)((float)voxel_idx - ((float)pt_grid_z * (float)voxel_grid_dim_x * (float)voxel_grid_dim_y) - ((float)pt_grid_y * (float)voxel_grid_dim_x));

  int search_radius = (int)round(trunc_margin / voxel_size);

  if (voxel_grid_occ[voxel_idx] > 0) {
    voxel_grid_TDF[voxel_idx] = 1.0f; // on surface
    return;
  }

  // Find closest surface point
  for (int iix = max(0, pt_grid_x - search_radius); iix < min(voxel_grid_dim_x, pt_grid_x + search_radius + 1); ++iix)
    for (int iiy = max(0, pt_grid_y - search_radius); iiy < min(voxel_grid_dim_y, pt_grid_y + search_radius + 1); ++iiy)
      for (int iiz = max(0, pt_grid_z - search_radius); iiz < min(voxel_grid_dim_z, pt_grid_z + search_radius + 1); ++iiz) {
        int iidx = iiz * voxel_grid_dim_x * voxel_grid_dim_y + iiy * voxel_grid_dim_x + iix;
        if (voxel_grid_occ[iidx] > 0) {
          float xd = (float)(pt_grid_x - iix);
          float yd = (float)(pt_grid_y - iiy);
          float zd = (float)(pt_grid_z - iiz);
          float dist = sqrtf(xd * xd + yd * yd + zd * zd) / (float)search_radius;
          if ((1.0f - dist) > voxel_grid_TDF[voxel_idx])
            voxel_grid_TDF[voxel_idx] = 1.0f - dist;
        }
      }
}

tdf_struct compute_tdf_grid(
    float truncated_margin,
    float voxel_size,
    int voxel_grid_padding,
    float * points,
    int num_pts
) {
  // Compute the minimum and maximum value from the points
  float voxel_grid_origin_x = points[0]; 
  float voxel_grid_origin_y = points[1]; 
  float voxel_grid_origin_z = points[2]; 
  float voxel_grid_max_x = points[0];
  float voxel_grid_max_y = points[1];
  float voxel_grid_max_z = points[2];
  for (int pt_idx = 0; pt_idx < num_pts; ++pt_idx) {
    voxel_grid_origin_x = min(voxel_grid_origin_x, points[pt_idx * 3 + 0]);
    voxel_grid_origin_y = min(voxel_grid_origin_y, points[pt_idx * 3 + 1]);
    voxel_grid_origin_z = min(voxel_grid_origin_z, points[pt_idx * 3 + 2]);
    voxel_grid_max_x = max(voxel_grid_max_x, points[pt_idx * 3 + 0]);
    voxel_grid_max_y = max(voxel_grid_max_y, points[pt_idx * 3 + 1]);
    voxel_grid_max_z = max(voxel_grid_max_z, points[pt_idx * 3 + 2]);
  }
  std::cout << "Initial voxel_grid_origin_x: " << voxel_grid_origin_x << std::endl;
  std::cout << "Initial voxel_grid_origin_y: " << voxel_grid_origin_y << std::endl;
  std::cout << "Initial voxel_grid_origin_z: " << voxel_grid_origin_z << std::endl;

  // Create a occupancy grid according to the maximum and minimum values of the point cloud
  int voxel_grid_dim_x = round((voxel_grid_max_x - voxel_grid_origin_x) / voxel_size) + 1 + voxel_grid_padding * 2;
  int voxel_grid_dim_y = round((voxel_grid_max_y - voxel_grid_origin_y) / voxel_size) + 1 + voxel_grid_padding * 2;
  int voxel_grid_dim_z = round((voxel_grid_max_z - voxel_grid_origin_z) / voxel_size) + 1 + voxel_grid_padding * 2;
  
  // Compute the minimum value (m) in each dimension after adding the voxel_grid_padding 
  voxel_grid_origin_x = voxel_grid_origin_x - voxel_grid_padding * voxel_size + voxel_size / 2;
  voxel_grid_origin_y = voxel_grid_origin_y - voxel_grid_padding * voxel_size + voxel_size / 2;
  voxel_grid_origin_z = voxel_grid_origin_z - voxel_grid_padding * voxel_size + voxel_size / 2;

  std::cout << "voxel_grid_origin_x: " << voxel_grid_origin_x << std::endl;
  std::cout << "voxel_grid_origin_y: " << voxel_grid_origin_y << std::endl;
  std::cout << "voxel_grid_origin_z: " << voxel_grid_origin_z << std::endl;

  std::cout << "Size of TDF voxel grid: " << voxel_grid_dim_x << 
               " x " << voxel_grid_dim_y << " x " << voxel_grid_dim_z << std::endl;
  std::cout << "Computing TDF voxel grid..." << std::endl;

  // Convert data to a voxel occupancy grid
  float * voxel_grid_occ = new float[voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z];
  // Initialize occupancy grid with 0s
  memset(voxel_grid_occ, 0, sizeof(float) * voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z);
  for (int pt_idx = 0; pt_idx < num_pts; ++pt_idx) {
    // Transform each point from meter to "voxel coordinates"
    int pt_grid_x = round((points[pt_idx * 3 + 0] - voxel_grid_origin_x) / voxel_size);
    int pt_grid_y = round((points[pt_idx * 3 + 1] - voxel_grid_origin_y) / voxel_size);
    int pt_grid_z = round((points[pt_idx * 3 + 2] - voxel_grid_origin_z) / voxel_size);
    // For each point in the point cloud assign it to a voxel in the occupancy
    // grid and set this voxel to be equal to one
    int v_idx = pt_grid_z * voxel_grid_dim_y * voxel_grid_dim_x + pt_grid_y * voxel_grid_dim_x + pt_grid_x;
    voxel_grid_occ[v_idx] = 1.0f;
  }

  // Initialize TDF voxel grid
  float * voxel_grid_TDF = new float[voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z];
  memset(voxel_grid_TDF, 0, sizeof(float) * voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z);

  // Copy voxel grids to GPU memory
  float * gpu_voxel_grid_occ;
  float * gpu_voxel_grid_TDF;
  hipMalloc(&gpu_voxel_grid_occ, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float));
  hipMalloc(&gpu_voxel_grid_TDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float));
  marvin::checkCUDA(__LINE__, hipGetLastError());
  hipMemcpy(
    gpu_voxel_grid_occ,
    voxel_grid_occ,
    voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float),
    hipMemcpyHostToDevice
  );
  hipMemcpy(
    gpu_voxel_grid_TDF,
    voxel_grid_TDF,
    voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float),
    hipMemcpyHostToDevice
  );
  marvin::checkCUDA(__LINE__, hipGetLastError());

  int CUDA_NUM_LOOPS = (int)ceil((float)(voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z) / (float)(CUDA_NUM_THREADS * CUDA_MAX_NUM_BLOCKS));

  for (int CUDA_LOOP_IDX = 0; CUDA_LOOP_IDX < CUDA_NUM_LOOPS; ++CUDA_LOOP_IDX) {
    ComputeTDF <<< CUDA_MAX_NUM_BLOCKS, CUDA_NUM_THREADS >>>(CUDA_LOOP_IDX, gpu_voxel_grid_occ, gpu_voxel_grid_TDF,
        voxel_grid_dim_x, voxel_grid_dim_y, voxel_grid_dim_z,
        voxel_grid_origin_x, voxel_grid_origin_y, voxel_grid_origin_z,
        voxel_size, truncated_margin);
  }

  // Load TDF voxel grid from GPU to CPU memory
  hipMemcpy(
    voxel_grid_TDF,
    gpu_voxel_grid_TDF,
    voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float),
    hipMemcpyDeviceToHost
  );
  marvin::checkCUDA(__LINE__, hipGetLastError());

  tdf_struct tdf;
  tdf.origin_x = voxel_grid_origin_x;
  tdf.origin_y = voxel_grid_origin_y;
  tdf.origin_z = voxel_grid_origin_z;
  tdf.dim_x = voxel_grid_dim_x;
  tdf.dim_y = voxel_grid_dim_y;
  tdf.dim_z = voxel_grid_dim_z;
  tdf.tdf_values = voxel_grid_TDF;

  return tdf;
}

void compute_random_keypoints(
    float * points,
    std::vector<int> random_idxs,
    float voxel_grid_origin_x,
    float voxel_grid_origin_y,
    float voxel_grid_origin_z,
    float voxel_size,
    float * keypts,
    float * keypts_grid
) {
    std::cout << "Finding random surface keypoints..." << std::endl;

    for (int keypt_idx = 0; keypt_idx < random_idxs.size(); ++keypt_idx) {
        keypts[keypt_idx * 3 + 0] = points[random_idxs[keypt_idx] * 3 + 0];
        keypts[keypt_idx * 3 + 1] = points[random_idxs[keypt_idx] * 3 + 1];
        keypts[keypt_idx * 3 + 2] = points[random_idxs[keypt_idx] * 3 + 2];
        keypts_grid[keypt_idx * 3 + 0] = round((points[keypt_idx * 3 + 0] - voxel_grid_origin_x) / voxel_size);
        keypts_grid[keypt_idx * 3 + 1] = round((points[keypt_idx * 3 + 1] - voxel_grid_origin_y) / voxel_size);
        keypts_grid[keypt_idx * 3 + 2] = round((points[keypt_idx * 3 + 2] - voxel_grid_origin_z) / voxel_size);
    }
}

int main(int argc, char *argv[]) {

  // Check if the command line arguments are correct
  if (argc != 7) {
    std::cout << "Usage: Generate 30x30x30 3D patch for each point in the pointcloud" << std::endl;
    std::cout << "reference_pointcloud: Input file containing the reference pointcloud to be processed" << std::endl;
    std::cout << "corresponding_pointcloud: Input file containing the corresponding pointcloud to be processed" << std::endl;
    std::cout << "non_matching_pointcloud: Input file containing the non matching pointcloud to be processed" << std::endl;
    std::cout << "output_prefix: Output prefix of the files used to store the computed descriptors and keypoints" << std::endl;
    std::cout << "voxel_size: Voxel size of the local 3D path " << std::endl;
    std::cout << "number_random_samples: The number of points to be sampled " << std::endl;
    return(1);
  }

  std::string reference_pointcloud_filename(argv[1]);
  std::string corresponding_pointcloud_filename(argv[2]);
  std::string non_matching_pointcloud_filename(argv[3]);
  std::string out_prefix_filename(argv[4]);
  float voxel_size = std::stof(argv[5]);
  int num_random_samples = std::atoi(argv[6]);
  int voxel_grid_padding = 15;
  float truncated_margin = voxel_size * 5;


  std::ifstream reference_pointcloud_file(reference_pointcloud_filename.c_str());
  if (!reference_pointcloud_file) {
    std::cerr << "Point cloud file not found." << std::endl;
    return -1;
  }
  int num_pts = 0;
  for (int line_idx = 0; line_idx < 7; ++line_idx) {
    std::string line_str;
    std::getline(reference_pointcloud_file, line_str);
    if (line_idx == 2) {
      std::istringstream tmp_line(line_str);
      std::string tmp_line_prefix;
      tmp_line >> tmp_line_prefix;
      tmp_line >> tmp_line_prefix;
      tmp_line >> num_pts;
    }
  }
  if (num_pts == 0) {
    std::cerr << "Third line of .ply file does not tell me number of points." << std::endl;
    return 0;
  }

  float * reference_points = new float[num_pts * 3]; // Nx3 matrix saved as float array (row-major order)
  if (IS_PLY_BINARY) {
    std::cout << "Reading point cloud in binary format..." << std::endl;
    reference_pointcloud_file.read((char*)reference_points, sizeof(float) * num_pts * 3);
  }
  else {
    std::cout << "Reading point cloud in ascii format..." << std::endl;
    // This is to read ply files that are in ascii format
    float ptx, pty, ptz;
    int i = 0;
    while (reference_pointcloud_file >> ptx >> pty >> ptz) {
      reference_points[i + 0] = ptx;
      reference_points[i + 1] = pty;
      reference_points[i + 2] = ptz;
      // std::cout << "ptx: " << ptx << " pty: " << pty << " ptz: " << ptz << std::endl;
      i += 3;
    }
  }
  reference_pointcloud_file.close();
  std::cout << "Loaded reference point cloud with " << num_pts << " points!" << std::endl;

  std::ifstream corresponding_pointcloud_file(corresponding_pointcloud_filename.c_str());
  if (!corresponding_pointcloud_file) {
    std::cerr << "Point cloud file not found." << std::endl;
    return -1;
  }
  int num_pts_corresponding = 0;
  for (int line_idx = 0; line_idx < 7; ++line_idx) {
    std::string line_str;
    std::getline(corresponding_pointcloud_file, line_str);
    if (line_idx == 2) {
      std::istringstream tmp_line(line_str);
      std::string tmp_line_prefix;
      tmp_line >> tmp_line_prefix;
      tmp_line >> tmp_line_prefix;
      tmp_line >> num_pts_corresponding;
    }
  }
  if (num_pts_corresponding == 0) {
    std::cerr << "Third line of .ply file does not tell me number of points." << std::endl;
    return 0;
  }
  
  float * corresponding_points = new float[num_pts_corresponding * 3]; // Nx3 matrix saved as float array (row-major order)
  if (IS_PLY_BINARY) {
    std::cout << "Reading corresponding point cloud in binary format..." << std::endl;
    corresponding_pointcloud_file.read((char*)corresponding_points, sizeof(float) * num_pts_corresponding * 3);
  }
  else {
    std::cout << "Reading corresponding point cloud in ascii format..." << std::endl;
    // This is to read ply files that are in ascii format
    float ptx, pty, ptz;
    int i = 0;
    while (corresponding_pointcloud_file >> ptx >> pty >> ptz) {
      corresponding_points[i + 0] = ptx;
      corresponding_points[i + 1] = pty;
      corresponding_points[i + 2] = ptz;
      // std::cout << "ptx: " << ptx << " pty: " << pty << " ptz: " << ptz << std::endl;
      i += 3;
    }
  }
  // This is to read ply files that are in binary format
  //pointcloud_file.read((char*)pts, sizeof(float) * num_pts * 3);
  corresponding_pointcloud_file.close();
  std::cout << "Loaded corresponding point cloud with " << num_pts_corresponding << " points!" << std::endl;

  std::ifstream non_matching_pointcloud_file(non_matching_pointcloud_filename.c_str());
  if (!non_matching_pointcloud_file) {
    std::cerr << "Point cloud file not found." << std::endl;
    return -1;
  }
  int num_pts_non_matching = 0;
  for (int line_idx = 0; line_idx < 7; ++line_idx) {
    std::string line_str;
    std::getline(non_matching_pointcloud_file, line_str);
    if (line_idx == 2) {
      std::istringstream tmp_line(line_str);
      std::string tmp_line_prefix;
      tmp_line >> tmp_line_prefix;
      tmp_line >> tmp_line_prefix;
      tmp_line >> num_pts_non_matching;
    }
  }
  if (num_pts_non_matching == 0) {
    std::cerr << "Third line of .ply file does not tell me number of points." << std::endl;
    return 0;
  }
  
  float * non_matching_points = new float[num_pts_non_matching * 3]; // Nx3 matrix saved as float array (row-major order)
  if (IS_PLY_BINARY) {
    std::cout << "Reading corresponding point cloud in binary format..." << std::endl;
    non_matching_pointcloud_file.read((char*)non_matching_points, sizeof(float) * num_pts_non_matching * 3);
  }
  else {
    std::cout << "Reading corresponding point cloud in ascii format..." << std::endl;
    // This is to read ply files that are in ascii format
    float ptx, pty, ptz;
    int i = 0;
    while (non_matching_pointcloud_file >> ptx >> pty >> ptz) {
     non_matching_points[i + 0] = ptx;
     non_matching_points[i + 1] = pty;
     non_matching_points[i + 2] = ptz;
      // std::cout << "ptx: " << ptx << " pty: " << pty << " ptz: " << ptz << std::endl;
      i += 3;
    }
  }
  // This is to read ply files that are in binary format
  //pointcloud_file.read((char*)pts, sizeof(float) * num_pts * 3);
  non_matching_pointcloud_file.close();
  std::cout << "Loaded non-matching point cloud with " << num_pts_non_matching << " points!" << std::endl;

  tdf_struct reference_tdf = compute_tdf_grid(
    truncated_margin,
    voxel_size,
    voxel_grid_padding,
    reference_points,
    num_pts
  );
  tdf_struct correspondence_tdf = compute_tdf_grid(
    truncated_margin,
    voxel_size,
    voxel_grid_padding,
    corresponding_points,
    num_pts_corresponding
  );
  tdf_struct non_matching_tdf = compute_tdf_grid(
    truncated_margin,
    voxel_size,
    voxel_grid_padding,
    non_matching_points,
    num_pts_non_matching
  );

  // Create a vector with matching indexes
  std::vector<int> matching_idxs;
  while (matching_idxs.size() < num_random_samples) {
    int idx = random_number(num_pts, 0);
    if (std::find( matching_idxs.begin(), matching_idxs.end(), idx ) == matching_idxs.end()) {
        matching_idxs.push_back(idx);
    }
  }

  // Create a vector with non matching indexes
  std::vector<int> non_matching_idxs;
  while (non_matching_idxs.size() < num_random_samples) {
    int idx = random_number(num_pts_non_matching, 0);
    if ( std::find( non_matching_idxs.begin(), non_matching_idxs.end(), idx ) == non_matching_idxs.end()) {
        non_matching_idxs.push_back(idx);
    }
  }

  float * keypts = new float[num_random_samples * 3];
  float * keypts_grid = new float[num_random_samples * 3];

  // Compute keypoints and the keypoints grid for the reference point cloud
  compute_random_keypoints(
    reference_points,
    matching_idxs,
    reference_tdf.origin_x,
    reference_tdf.origin_y,
    reference_tdf.origin_z,
    voxel_size,
    keypts,
    keypts_grid
  );

  // Save keypoints as binary file (Nx30x30 float array, row-major order)
  std::string p1_saveto_path = out_prefix_filename + ".p1_tdf.bin";
  std::ofstream p1_out_file(p1_saveto_path, std::ios::binary | std::ios::app);

  // Compute the 30x30x30 value of sampled keypoints
  for ( int keypt_idx = 0; keypt_idx < num_random_samples; ++keypt_idx) {
    float keypt_grid_x = keypts_grid[keypt_idx * 3 + 0];
    float keypt_grid_y = keypts_grid[keypt_idx * 3 + 1];
    float keypt_grid_z = keypts_grid[keypt_idx * 3 + 2];

    // Get local TDF around keypoint
    float * local_voxel_grid_TDF = new float[30 * 30 * 30];
    int local_voxel_idx = 0;
    for (int z = keypt_grid_z - 15; z < keypt_grid_z + 15; ++z)
        for (int y = keypt_grid_y - 15; y < keypt_grid_y + 15; ++y)
            for (int x = keypt_grid_x - 15; x < keypt_grid_x + 15; ++x) {
                local_voxel_grid_TDF[ local_voxel_idx ] = 
         reference_tdf.tdf_values[ z * reference_tdf.dim_x * reference_tdf.dim_y + y * reference_tdf.dim_x + x ];
                local_voxel_idx++;
          }

    std::cout << "Saving TDF values for the " << keypt_idx <<" keypoint " << "from the " << num_random_samples << " to disk (.p1_tdf.bin)..." << std::endl;
    p1_out_file.write((char*)local_voxel_grid_TDF, sizeof(float)*30*30*30);

    delete [] local_voxel_grid_TDF;
 }
 p1_out_file.close();

 // Compute keypoints and the keypoints grid for the reference point cloud
 compute_random_keypoints(
    corresponding_points,
    matching_idxs,
    correspondence_tdf.origin_x,
    correspondence_tdf.origin_y,
    correspondence_tdf.origin_z,
    voxel_size,
    keypts,
    keypts_grid
 );

 // Save keypoints as binary file (Nx30x30 float array, row-major order)
 std::string p2_saveto_path = out_prefix_filename + ".p2_tdf.bin";
 std::ofstream p2_out_file(p2_saveto_path, std::ios::binary | std::ios::app);

 // Compute the 30x30x30 value of sampled keypoints
 for ( int keypt_idx = 0; keypt_idx < num_random_samples; ++keypt_idx) {
    float keypt_grid_x = keypts_grid[keypt_idx * 3 + 0];
    float keypt_grid_y = keypts_grid[keypt_idx * 3 + 1];
    float keypt_grid_z = keypts_grid[keypt_idx * 3 + 2];

    // Get local TDF around keypoint
    float * local_voxel_grid_TDF = new float[30 * 30 * 30];
    int local_voxel_idx = 0;
    for (int z = keypt_grid_z - 15; z < keypt_grid_z + 15; ++z)
        for (int y = keypt_grid_y - 15; y < keypt_grid_y + 15; ++y)
            for (int x = keypt_grid_x - 15; x < keypt_grid_x + 15; ++x) {
                local_voxel_grid_TDF[ local_voxel_idx ] = 
        correspondence_tdf.tdf_values[ z * correspondence_tdf.dim_x * correspondence_tdf.dim_y + y * correspondence_tdf.dim_x + x ];
                local_voxel_idx++;
          }

    std::cout << "Saving TDF values for the " << keypt_idx <<" keypoint " << "from the " << num_random_samples << " to disk (.p2_tdf.bin)..." << std::endl;
    p2_out_file.write((char*)local_voxel_grid_TDF, sizeof(float)*30*30*30);

    delete [] local_voxel_grid_TDF;
 }
 p2_out_file.close();

 // Compute keypoints and the keypoints grid for the reference point cloud
 compute_random_keypoints(
    non_matching_points,
    non_matching_idxs,
    non_matching_tdf.origin_x,
    non_matching_tdf.origin_y,
    non_matching_tdf.origin_z,
    voxel_size,
    keypts,
    keypts_grid
 );

 // Save keypoints as binary file (Nx30x30 float array, row-major order)
 std::string p3_saveto_path = out_prefix_filename + ".p3_tdf.bin";
 std::ofstream p3_out_file(p3_saveto_path, std::ios::binary | std::ios::app);

 // Compute the 30x30x30 value of sampled keypoints
 for ( int keypt_idx = 0; keypt_idx < num_random_samples; ++keypt_idx) {
    float keypt_grid_x = keypts_grid[keypt_idx * 3 + 0];
    float keypt_grid_y = keypts_grid[keypt_idx * 3 + 1];
    float keypt_grid_z = keypts_grid[keypt_idx * 3 + 2];

    // Get local TDF around keypoint
    float * local_voxel_grid_TDF = new float[30 * 30 * 30];
    int local_voxel_idx = 0;
    for (int z = keypt_grid_z - 15; z < keypt_grid_z + 15; ++z)
        for (int y = keypt_grid_y - 15; y < keypt_grid_y + 15; ++y)
            for (int x = keypt_grid_x - 15; x < keypt_grid_x + 15; ++x) {
                local_voxel_grid_TDF[ local_voxel_idx ] = 
        non_matching_tdf.tdf_values[ z * non_matching_tdf.dim_x * non_matching_tdf.dim_y + y * non_matching_tdf.dim_x + x ];
                local_voxel_idx++;
          }

    std::cout << "Saving TDF values for the " << keypt_idx <<" keypoint " << "from the " << num_random_samples << " to disk (.p3_tdf.bin)..." << std::endl;
    p3_out_file.write((char*)local_voxel_grid_TDF, sizeof(float)*30*30*30);

    delete [] local_voxel_grid_TDF;
 }
 p3_out_file.close();
 delete [] keypts;
 delete [] keypts_grid;
}
